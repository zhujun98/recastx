/**
 * Copyright (c) Paul Scherrer Institut
 * Author: Jun Zhu
 *
 * Distributed under the terms of the BSD 3-Clause License.
 *
 * The full license is in the file LICENSE, distributed with this software.
*/
#include <boost/shared_ptr.hpp>

#include "recon/cuda/utils.cuh"
#include "recon/cuda/recon3d.cuh"
#include "recon/cuda/memory.cuh"
#include "recon/cuda/stream.cuh"

namespace recastx::recon {

AstraReconstructable::AstraReconstructable(const VolumeGeometry& geom)
        : geom_(new astra::CVolumeGeometry3D(
                    geom.col_count, geom.row_count, geom.slice_count,
                    geom.min_x, geom.min_y, geom.min_z, geom.max_x, geom.max_y, geom.max_z)),
          mem_(new AstraMemHandle(geom.col_count, geom.row_count, geom.slice_count)),
          data_(new astra::CFloat32VolumeData3DGPU(geom_.get(), mem_->handle())),
          stream_(new Stream) {

    spdlog::info("[Init] - Volume geometry: shape {} x {} x {}, x [{}, {}], y [{}, {}], z [{}, {}]",
                 geom_->getGridRowCount(), geom_->getGridColCount(), geom_->getGridSliceCount(),
                 geom_->getWindowMinX(), geom_->getWindowMaxX(),
                 geom_->getWindowMinY(), geom_->getWindowMaxY(),
                 geom_->getWindowMinZ(), geom_->getWindowMaxZ());
}

AstraReconstructable::~AstraReconstructable() = default;

void AstraReconstructable::copySlice(float* buffer) {
    unsigned int x = geom_->getGridColCount();
    unsigned int y = geom_->getGridRowCount();
    assert(x == y);
    copyFromDevice(buffer, mem_.get(), x, y, 1, x);
}

void AstraReconstructable::copyVolume(float* buffer) {
    unsigned int x = geom_->getGridColCount();
    unsigned int y = geom_->getGridRowCount();
    unsigned int z = geom_->getGridSliceCount();
    assert(x == y);
    assert(x == z);
    copyFromDevice(buffer, mem_.get(), x, y, z, x);
}

bool AstraReconstructable::copyFromDevice(float* dst, const AstraMemHandle* src,
                                          unsigned int x, unsigned int y, unsigned int z, unsigned int pitch) {
    spdlog::debug("Copying {} x {} x {} from GPU", x, y, z);

    const auto& handle = src->handle();
    hipMemcpy3DParms p;

    p.srcPos = make_hipPos(0, 0, 0);
    if (handle.d->ptr.ptr != nullptr) {
        p.srcArray = nullptr;
        p.srcPtr = handle.d->ptr;
        p.extent = make_hipExtent(x * sizeof(float), y, z);
    } else {
        p.srcArray = handle.d->arr;
        p.srcPtr = make_hipPitchedPtr(nullptr, 0, 0, 0);
        p.extent = make_hipExtent(x, y, z);
    }

    p.dstArray = nullptr;
    p.dstPos = make_hipPos(0 * sizeof(float), 0, 0);
    p.dstPtr = make_hipPitchedPtr((void*)dst, pitch * sizeof(float), x * sizeof(float), y);

    p.kind = hipMemcpyDeviceToHost;

    return checkCudaError(hipMemcpy3DAsync(&p, stream_->d));
}

} // namespace recastx::recon