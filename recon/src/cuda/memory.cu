/**
 * Copyright (c) Paul Scherrer Institut
 * Author: Jun Zhu
 *
 * Distributed under the terms of the BSD 3-Clause License.
 *
 * The full license is in the file LICENSE, distributed with this software.
*/
#include "spdlog/spdlog.h"

#include "recon/cuda/memory.cuh"
#include "recon/cuda/utils.cuh"
#include "recon/cuda/stream.cuh"

namespace recastx::recon {

AstraMemHandleBase::AstraMemHandleBase() = default;

AstraMemHandleBase::~AstraMemHandleBase() = default;

AstraMemHandleBase::AstraMemHandleBase(AstraMemHandleBase&& other) noexcept : handle_(std::move(other.handle_)) {
}

AstraMemHandleBase& AstraMemHandleBase::operator=(AstraMemHandleBase&& other) noexcept {
    handle_ = std::move(other.handle_);
    return *this;
}

unsigned int AstraMemHandleBase::x() const { return handle_.d->nx; }
unsigned int AstraMemHandleBase::y() const { return handle_.d->ny; }
unsigned int AstraMemHandleBase::z() const { return handle_.d->nz; }
unsigned int AstraMemHandleBase::size() const { return handle_.d->nx * handle_.d->ny * handle_.d->nz;; }


AstraMemHandle::AstraMemHandle(unsigned int x, unsigned int y, unsigned int z) : AstraMemHandleBase() {
    handle_.d = boost::shared_ptr<astraCUDA3d::SMemHandle3D_internal>(new astraCUDA3d::SMemHandle3D_internal);
    handle_.d->arr = nullptr;

    if (!checkCudaError(hipMalloc3D(&handle_.d->ptr, make_hipExtent(sizeof(float) * x, y, z)))) {
        throw std::runtime_error("Failed to allocate a 3D logical array");
    }

    handle_.d->nx = x;
    handle_.d->ny = y;
    handle_.d->nz = z;

    auto [gpu_memory_available, gpu_memory_total] = queryGPUMemory();
    spdlog::debug("Allocated {} x {} x {} on GPU. ({} / {})", x, y, z, gpu_memory_available, gpu_memory_total);
}

AstraMemHandle::~AstraMemHandle() {
    if (handle_.d != nullptr) checkCudaError(hipFree(handle_.d->ptr.ptr));
}

AstraMemHandle::AstraMemHandle(AstraMemHandle&& other) noexcept = default;

AstraMemHandle& AstraMemHandle::operator=(AstraMemHandle&& other) noexcept = default;

AstraMemHandleArray::AstraMemHandleArray(unsigned int x, unsigned int y, unsigned int z) : AstraMemHandleBase(){
    handle_.d = boost::shared_ptr<astraCUDA3d::SMemHandle3D_internal>(new astraCUDA3d::SMemHandle3D_internal);
    handle_.d->ptr = make_hipPitchedPtr(nullptr, 0, 0, 0);

    auto channel_desc = hipCreateChannelDesc<float>();
    auto extent = make_hipExtent(x, y, z);
    if (!checkCudaError(hipMalloc3DArray(&handle_.d->arr, &channel_desc, extent))) {
        throw std::runtime_error("Failed to allocate a 3D array");
    }

    handle_.d->nx = x;
    handle_.d->ny = y;
    handle_.d->nz = z;

    auto [gpu_memory_available, gpu_memory_total] = queryGPUMemory();
    spdlog::debug("Allocated {} x {} x {} on GPU. ({} / {})", x, y, z, gpu_memory_available, gpu_memory_total);
}

AstraMemHandleArray::~AstraMemHandleArray() {
    if (handle_.d != nullptr) checkCudaError(hipFreeArray(handle_.d->arr));
}

AstraMemHandleArray::AstraMemHandleArray(AstraMemHandleArray&& other) noexcept = default;

AstraMemHandleArray& AstraMemHandleArray::operator=(AstraMemHandleArray&& other) noexcept = default;

} // recastx::recon