/**
 * Copyright (c) Paul Scherrer Institut
 * Author: Jun Zhu
 *
 * Distributed under the terms of the BSD 3-Clause License.
 *
 * The full license is in the file LICENSE, distributed with this software.
*/
#include "spdlog/spdlog.h"

#include "recon/cuda/sinogram_proxy.cuh"
#include "recon/cuda/utils.cuh"
#include "recon/cuda/stream.cuh"

namespace recastx::recon {

SinogramProxy::SinogramProxy(size_t group_size)
    : start_{0}, group_size_{group_size}, stream_(new Stream) {
}

SinogramProxy::~SinogramProxy() = default;

void SinogramProxy::copyToDevice(astra::CFloat32ProjectionData3DGPU *dst) {
    const float *src = buffer_.front().data();
    size_t count = buffer_.front().shape()[0];
    size_t start = start_;
    size_t end = (start + count - 1) % group_size_;

    if (end > start) {
        copyToDevice(dst, src, start, end);
        spdlog::debug("Uploaded sinograms {} - {}", start, end);
    } else {
        copyToDevice(dst, src, start, group_size_ - 1);
        spdlog::debug("Uploaded sinograms {} - {}", start, group_size_ - 1);
        copyToDevice(dst, src, 0, end);
        spdlog::debug("Uploaded sinograms {} - {}", 0, end);
    }

    start_ = (end + 1) % group_size_;
}

void SinogramProxy::copyToDevice(astra::CFloat32ProjectionData3DGPU* proj,
                                 const float* data, unsigned int start, unsigned int end) {
    unsigned int x = proj->getDetectorColCount();
    unsigned int y = end - start + 1;
    unsigned int z = proj->getDetectorRowCount();

    hipMemcpy3DParms p;
    p.srcArray = nullptr;
    p.srcPos = make_hipPos(0, 0, 0);
    p.srcPtr = make_hipPitchedPtr((void *) data, x * sizeof(float), x, y);
    assert(proj->getHandle().d->arr != nullptr);
    p.dstArray = proj->getHandle().d->arr;
    p.dstPos = make_hipPos(0, start, 0);
    p.dstPtr = make_hipPitchedPtr(nullptr, 0, 0, 0);;
    p.extent = make_hipExtent(x, y, z);
    p.kind = hipMemcpyHostToDevice;

    if (!checkCudaError(hipMemcpy3DAsync(&p, stream_->d))) {
        spdlog::error("Failed to copy sinogram data ({} - {}) from CPU to GPU", start, end);
    }
}

} // recastx::recon