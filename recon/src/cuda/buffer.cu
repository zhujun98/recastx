/**
 * Copyright (c) Paul Scherrer Institut
 * Author: Jun Zhu
 *
 * Distributed under the terms of the BSD 3-Clause License.
 *
 * The full license is in the file LICENSE, distributed with this software.
*/
#include <functional>

#include <recon/cuda/utils.cuh>
#include "recon/cuda/buffer.cuh"

namespace recastx::recon {

template<typename T, size_t N>
DeviceTensor<T, N>::DeviceTensor() = default;

template<typename T, size_t N>
DeviceTensor<T, N>::~DeviceTensor() {
    if (data_ != nullptr) hipHostFree(data_);
}

template<typename T, size_t N>
void DeviceTensor<T, N>::swap(DeviceTensor& other) noexcept {
    std::swap(data_, other.data_);
    shape_.swap(other.shape_);
}

template<typename T, size_t N>
void DeviceTensor<T, N>::resize(const ShapeType& shape) {
    if (data_ != nullptr) hipHostFree(data_);
    size_t n = std::accumulate(std::begin(shape), std::end(shape), 1, std::multiplies<size_t>());
    checkCudaError(hipHostMalloc((void**)&data_, n * sizeof(ValueType)));
    shape_ = shape;
}

template class DeviceTensor<ProDtype, 3>;
template class DeviceTensor<ProDtype, 2>;


TripleGpuTensorBuffer::TripleGpuTensorBuffer() = default;

TripleGpuTensorBuffer::~TripleGpuTensorBuffer() = default;

void TripleGpuTensorBuffer::resize(const ShapeType& shape) {
    std::lock_guard lk(this->mtx_);
    this->back_.resize(shape);
    this->ready_.resize(shape);
    this->front_.resize(shape);
}

} // recastx::recon