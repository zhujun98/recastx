/**
 * Copyright (c) Paul Scherrer Institut
 * Author: Jun Zhu
 *
 * Distributed under the terms of the BSD 3-Clause License.
 *
 * The full license is in the file LICENSE, distributed with this software.
*/
#include <recon/cuda/utils.cuh>
#include "recon/cuda/buffer.cuh"

namespace recastx::recon {

DeviceTensor::DeviceTensor() = default;

DeviceTensor::~DeviceTensor() {
    if (data_ != nullptr) hipHostFree(data_);
}

void DeviceTensor::swap(DeviceTensor& other) noexcept {
    std::swap(data_, other.data_);
    shape_.swap(other.shape_);
}

void DeviceTensor::resize(const ShapeType& shape) {
    if (data_ != nullptr) hipHostFree(data_);
    checkCudaError(hipHostMalloc((void**)&data_, shape[0] * shape[1] * shape[2] * sizeof(ValueType)));
    shape_ = shape;
}

void TripleGpuTensorBuffer::resize(const ShapeType& shape) {
    std::lock_guard(this->mtx_);
    this->back_.resize(shape);
    this->ready_.resize(shape);
    this->front_.resize(shape);
}

} // recastx::recon