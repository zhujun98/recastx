/**
 * Copyright (c) Paul Scherrer Institut
 * Author: Jun Zhu
 *
 * Distributed under the terms of the BSD 3-Clause License.
 *
 * The full license is in the file LICENSE, distributed with this software.
*/
#include "spdlog/spdlog.h"

#include "recon/cuda/sinogram_loader.cuh"
#include "recon/cuda/utils.cuh"
#include "recon/cuda/stream.cuh"

namespace recastx::recon {

SinogramLoader::SinogramLoader(size_t group_size)
    : start_{0}, group_size_{group_size}, stream_(new Stream) {
}

SinogramLoader::~SinogramLoader() = default;

void SinogramLoader::load(astra::CFloat32ProjectionData3DGPU *dst, const float *src, size_t count) {
    size_t start = start_;
    size_t end = (start + count - 1) % group_size_;

    if (end > start) {
        copyToDevice(dst, src, start, end);
        spdlog::debug("Uploaded sinograms {} - {}", start, end);
    } else {
        copyToDevice(dst, src, start, group_size_ - 1);
        spdlog::debug("Uploaded sinograms {} - {}", start, group_size_ - 1);
        copyToDevice(dst, src, 0, end);
        spdlog::debug("Uploaded sinograms {} - {}", 0, end);
    }

    start_ = (end + 1) % group_size_;
}

void SinogramLoader::copyToDevice(astra::CFloat32ProjectionData3DGPU* proj,
                                  const float* data, unsigned int start, unsigned int end) {
    unsigned int x = proj->getDetectorColCount();
    unsigned int y = end - start + 1;
    unsigned int z = proj->getDetectorRowCount();

    hipMemcpy3DParms p;
    p.srcArray = nullptr;
    p.srcPos = make_hipPos(0, 0, 0);
    p.srcPtr = make_hipPitchedPtr((void *) data, x * sizeof(float), x, y);
    assert(proj->getHandle().d->arr != nullptr);
    p.dstArray = proj->getHandle().d->arr;
    p.dstPos = make_hipPos(0, start, 0);
    p.dstPtr = make_hipPitchedPtr(nullptr, 0, 0, 0);;
    p.extent = make_hipExtent(x, y, z);
    p.kind = hipMemcpyHostToDevice;

    if (!checkCudaError(hipMemcpy3DAsync(&p, stream_->d))) {
        spdlog::error("Failed to copy sinogram data ({} - {}) from CPU to GPU", start, end);
    }
}

} // recastx::recon